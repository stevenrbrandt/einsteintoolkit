#include "hip/hip_runtime.h"
#include <cctk.h>
#include <cctk_Arguments.h>

#include <cstdarg>
#include <cstdio>
#include <cstdlib>
#include <cstring>
using namespace std;



namespace HelloWorldCUDA {
  
  // Check a return value, and if there is an error, output a
  // human-readable error message
  void check_error(hipError_t cerr, char const *msg = "", ...)
#ifdef __GNUC__
    __attribute__((format (printf, 2, 3)))
#endif
    ;
  void check_error(hipError_t cerr, char const *msg, ...)
  {
    if (cerr) {
      if (strcmp(msg, "")) {
        va_list ap;
        va_start(ap, msg);
        char *usermsg;
        vasprintf(&usermsg, msg, ap);
        va_end(ap);
        CCTK_VWarn(CCTK_WARN_ABORT, __LINE__, __FILE__, CCTK_THORNSTRING,
                   "CUDA Error %d: %s:\n%s",
                   int(cerr), hipGetErrorString(cerr), usermsg);
        free(usermsg);
      } else {
        CCTK_VWarn(CCTK_WARN_ABORT, __LINE__, __FILE__, CCTK_THORNSTRING,
                   "CUDA Error %d: %s\n",
                   int(cerr), hipGetErrorString(cerr));
      }
    }
  }



  extern "C"
  void HelloWorldCUDA_initial(CCTK_ARGUMENTS)
  {
    // Output device properties
    hipDeviceProp_t prop;
    hipError_t cerr = hipGetDeviceProperties(&prop, 0);
    check_error(cerr, "Could not get device properties");
    printf("CUDA device properties (device %d):\n", 0);
    printf("   name:                        %s\n", prop.name);
    printf("   totalGlobalMem:              %zu\n", prop.totalGlobalMem);
    printf("   sharedMemPerBlock:           %zu\n", prop.sharedMemPerBlock);
    printf("   regsPerBlock:                %d\n", prop.regsPerBlock);
    printf("   warpSize:                    %d\n", prop.warpSize);
    printf("   memPitch:                    %zu\n", prop.memPitch);
    printf("   maxThreadsPerBlock:          %d\n", prop. maxThreadsPerBlock);
    printf("   maxThreadsDim:               %d %d %d\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
    printf("   maxGridSize:                 %d %d %d\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
    printf("   clockRate:                   %d\n", prop.clockRate);
    printf("   totalConstMem                %zu\n", prop.totalConstMem);
    printf("   major:                       %d\n", prop.major);
    printf("   minor:                       %d\n", prop.minor);
    printf("   textureAlignment:            %zu\n", prop.textureAlignment);
    printf("   deviceOverlap:               %d\n", prop.deviceOverlap);
    printf("   multiProcessorCount:         %d\n", prop.multiProcessorCount);
    printf("   kernelExecTimeoutEnabled:    %d\n", prop.kernelExecTimeoutEnabled);
    printf("   integrated:                  %d\n", prop.integrated);
    printf("   canMapHostMemory:            %d\n", prop.canMapHostMemory);
    printf("   computeMode:                 %d\n", prop.computeMode);
    printf("   maxTexture1D:                %d\n", prop.maxTexture1D);
    printf("   maxTexture2D:                %d %d\n", prop.maxTexture2D[0], prop.maxTexture2D[1]);
    printf("   maxTexture3D:                %d %d %d\n", prop.maxTexture3D[0], prop.maxTexture3D[1], prop.maxTexture3D[2]);
    printf("   maxTexture1DLayered:         %d %d\n", prop.maxTexture1DLayered[0], prop.maxTexture1DLayered[1]);
    printf("   maxTexture2DLayered:         %d %d %d\n", prop.maxTexture2DLayered[0], prop.maxTexture2DLayered[1], prop.maxTexture2DLayered[2]);
    printf("   surfaceAlignment:            %zu\n", prop.surfaceAlignment);
    printf("   concurrentKernels:           %d\n", prop.concurrentKernels);
    printf("   ECCEnabled:                  %d\n", prop.ECCEnabled);
    printf("   pciBusID:                    %d\n", prop.pciBusID);
    printf("   pciDeviceID:                 %d\n", prop.pciDeviceID);
    printf("   pciDomainID:                 %d\n", prop.pciDomainID);
    printf("   tccDriver:                   %d\n", prop.tccDriver);
    printf("   asyncEngineCount:            %d\n", prop.asyncEngineCount);
    printf("   unifiedAddressing:           %d\n", prop.unifiedAddressing);
    printf("   memoryClockRate:             %d\n", prop.memoryClockRate);
    printf("   memoryBusWidth:              %d\n", prop.memoryBusWidth);
    printf("   l2CacheSize:                 %d\n", prop.l2CacheSize);
    printf("   maxThreadsPerMultiProcessor: %d\n", prop.maxThreadsPerMultiProcessor);
  }
  
  
  
  __global__ void add(const int val1, const int val2, int* res)
  {
    const size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    const size_t j = blockIdx.y * blockDim.y + threadIdx.y;
    const size_t k = blockIdx.z * blockDim.z + threadIdx.z;
    if (i==0 && j==0 && k==0) {
      *res = val1 + val2;
    }
  }
  
  extern "C"
  void HelloWorldCUDA_evol(CCTK_ARGUMENTS)
  {
    DECLARE_CCTK_ARGUMENTS;
    
    const int val1 = cctk_iteration;
    const int val2 = 3;
    int res = 42;                 // poison
    
    const dim3 blockDim(4, 4, 4);
    const dim3 gridDim((cctk_lsh[0] + blockDim.x - 1) / blockDim.x,
                       (cctk_lsh[1] + blockDim.y - 1) / blockDim.y,
                       (cctk_lsh[2] + blockDim.z - 1) / blockDim.z);
    add<<<gridDim, blockDim>>>(val1, val2, &res);
    
    CCTK_VInfo(CCTK_THORNSTRING, "CUDA says: %d + %d = %d", val1, val2, res);
  }
  
} // namespace HelloWorldCUDA
