#include "hip/hip_runtime.h"
// -*-C++-*-

#include <assert.h>
#include <stdarg.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include <hip/hip_runtime.h>

#include <cctk.h>
#include <cctk_Arguments.h>
#include <cctk_Parameters.h>



namespace WaveToyCUDA {
  
  // Check a return value, and if there is an error, output a
  // human-readable error message
  void check_error(hipError_t cerr, char const *msg = "", ...)
#ifdef __GNUC__
  __attribute__((format (printf, 2, 3)))
#endif
  ;
  void check_error(hipError_t cerr, char const *msg, ...)
  {
    if (cerr) {
      if (strcmp(msg, "")) {
        va_list ap;
        va_start(ap, msg);
        char *usermsg;
        vasprintf(&usermsg, msg, ap);
        va_end(ap);
        CCTK_VWarn(CCTK_WARN_ABORT, __LINE__, __FILE__, CCTK_THORNSTRING,
                   "CUDA Error %d: %s:\n%s",
                   (int)cerr, hipGetErrorString(cerr), usermsg);
        free(usermsg);
      } else {
        CCTK_VWarn(CCTK_WARN_ABORT, __LINE__, __FILE__, CCTK_THORNSTRING,
                   "CUDA Error %d: %s\n",
                   (int)cerr, hipGetErrorString(cerr));
      }
    }
  }
  
  
  
  // Access a grid function in a kernel
  __device__ CCTK_REAL& gfelt(hipPitchedPtr const& u,
                              size_t i, size_t j, size_t k)
  {
    return *(CCTK_REAL*)&((char*)u.ptr)[i*sizeof(CCTK_REAL) +
                                        j*u.pitch +
                                        k*u.pitch*u.ysize];
  }
  
  // Data living in the device memory
  namespace dev {
    hipExtent ext;
    hipPitchedPtr u;
  } // namespace dev
  
  
  
  // A simple kernel
  __global__ void init(size_t const lsh0, size_t const lsh1, size_t const lsh2,
                       hipPitchedPtr const u)
  {
    size_t const i = blockIdx.x * blockDim.x + threadIdx.x;
    size_t const j = blockIdx.y * blockDim.y + threadIdx.y;
    size_t const k = blockIdx.z * blockDim.z + threadIdx.z;
    if (i<lsh0 && j<lsh1 && k<lsh2) {
      gfelt(u,i,j,k) = 0.0;
    }
  }
  
  
  
  extern "C"
  void WaveToyCUDA_Init(CCTK_ARGUMENTS)
  {
    DECLARE_CCTK_ARGUMENTS;
    DECLARE_CCTK_PARAMETERS;
    
    hipError_t cerr;
    
    
    
    // Output device properties
    hipDeviceProp_t prop;
    cerr = hipGetDeviceProperties(&prop, 0);
    check_error(cerr, "Could not get device properties");
    printf("CUDA device properties (device %d):\n", 0);
    printf("   name:                        %s\n", prop.name);
    printf("   totalGlobalMem:              %zu\n", prop.totalGlobalMem);
    printf("   sharedMemPerBlock:           %zu\n", prop.sharedMemPerBlock);
    printf("   regsPerBlock:                %d\n", prop.regsPerBlock);
    printf("   warpSize:                    %d\n", prop.warpSize);
    printf("   memPitch:                    %zu\n", prop.memPitch);
    printf("   maxThreadsPerBlock:          %d\n", prop. maxThreadsPerBlock);
    printf("   maxThreadsDim:               %d %d %d\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
    printf("   maxGridSize:                 %d %d %d\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
    printf("   clockRate:                   %d\n", prop.clockRate);
    printf("   totalConstMem                %zu\n", prop.totalConstMem);
    printf("   major:                       %d\n", prop.major);
    printf("   minor:                       %d\n", prop.minor);
    printf("   textureAlignment:            %zu\n", prop.textureAlignment);
    printf("   deviceOverlap:               %d\n", prop.deviceOverlap);
    printf("   multiProcessorCount:         %d\n", prop.multiProcessorCount);
    printf("   kernelExecTimeoutEnabled:    %d\n", prop.kernelExecTimeoutEnabled);
    printf("   integrated:                  %d\n", prop.integrated);
    printf("   canMapHostMemory:            %d\n", prop.canMapHostMemory);
    printf("   computeMode:                 %d\n", prop.computeMode);
    printf("   maxTexture1D:                %d\n", prop.maxTexture1D);
    printf("   maxTexture2D:                %d %d\n", prop.maxTexture2D[0], prop.maxTexture2D[1]);
    printf("   maxTexture3D:                %d %d %d\n", prop.maxTexture3D[0], prop.maxTexture3D[1], prop.maxTexture3D[2]);
    printf("   maxTexture1DLayered:         %d %d\n", prop.maxTexture1DLayered[0], prop.maxTexture1DLayered[1]);
    printf("   maxTexture2DLayered:         %d %d %d\n", prop.maxTexture2DLayered[0], prop.maxTexture2DLayered[1], prop.maxTexture2DLayered[2]);
    printf("   surfaceAlignment:            %zu\n", prop.surfaceAlignment);
    printf("   concurrentKernels:           %d\n", prop.concurrentKernels);
    printf("   ECCEnabled:                  %d\n", prop.ECCEnabled);
    printf("   pciBusID:                    %d\n", prop.pciBusID);
    printf("   pciDeviceID:                 %d\n", prop.pciDeviceID);
    printf("   pciDomainID:                 %d\n", prop.pciDomainID);
    printf("   tccDriver:                   %d\n", prop.tccDriver);
    printf("   asyncEngineCount:            %d\n", prop.asyncEngineCount);
    printf("   unifiedAddressing:           %d\n", prop.unifiedAddressing);
    printf("   memoryClockRate:             %d\n", prop.memoryClockRate);
    printf("   memoryBusWidth:              %d\n", prop.memoryBusWidth);
    printf("   l2CacheSize:                 %d\n", prop.l2CacheSize);
    printf("   maxThreadsPerMultiProcessor: %d\n", prop.maxThreadsPerMultiProcessor);
    
    
    
    // Allocate memory
    if (!dev::u.ptr) {
      dev::ext = make_hipExtent
        (sizeof(CCTK_REAL)*cctk_lsh[0], cctk_lsh[1], cctk_lsh[2]);
      cerr = hipMalloc3D(&dev::u, dev::ext);
      check_error(cerr, "Failed to allocate [%d,%d,%d] array",
                  cctk_lsh[0], cctk_lsh[1], cctk_lsh[2]);
    }
    
    // Call a kernel
    dim3 const blockDim(4, 4, 4);
    dim3 const gridDim((cctk_lsh[0] + blockDim.x - 1) / blockDim.x,
                       (cctk_lsh[1] + blockDim.y - 1) / blockDim.y,
                       (cctk_lsh[2] + blockDim.z - 1) / blockDim.z);
    init<<<gridDim, blockDim>>>(cctk_lsh[0], cctk_lsh[1], cctk_lsh[2],
                                dev::u);
    cerr = hipGetLastError();
    check_error(cerr, "Could not call routine \"init\"");
    
    // Copy data to host
    hipMemcpy3DParms parms = {0};
    parms.srcPtr = dev::u;
    parms.dstPtr = make_hipPitchedPtr
      (u, sizeof(CCTK_REAL)*cctk_ash[0], cctk_lsh[0], cctk_lsh[1]);
    parms.extent = dev::ext;
    parms.kind = hipMemcpyDeviceToHost;
    cerr = hipMemcpy3D(&parms);
    check_error(cerr, "Failed to copy [%d,%d,%d] array",
                cctk_lsh[0], cctk_lsh[1], cctk_lsh[2]);
    
    // Output grid function
    // for (ptrdiff_t k=0; k<cctk_lsh[2]; ++k) {
    //   for (ptrdiff_t j=0; j<cctk_lsh[1]; ++j) {
    //     for (ptrdiff_t i=0; i<cctk_lsh[0]; ++i) {
    //       ptrdiff_t const ind3d = CCTK_GFINDEX3D(cctkGH, i,j,k);
    //       printf("[%td,%td,%td]=%g\n", i,j,k, (double)u[ind3d]);
    //     }
    //   }
    // }
  }
  
} // namespace WaveToyCUDA
